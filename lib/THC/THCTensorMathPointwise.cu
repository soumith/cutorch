#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCTensorRandom.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC)                   \
  struct Tensor##NAME##Op {                                             \
    __device__ __forceinline__ void operator()(float* out, float* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(float* v) const {        \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCudaTensor_##NAME(THCState* state, THCudaTensor* self_, THCudaTensor* src) { \
    if (self_ == src) {                                                 \
      if (!THCudaTensor_pointwiseApply1(state, self_, Tensor##NAME##Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING); \
      }                                                                 \
    } else {                                                            \
      THCudaTensor_resizeAs(state, self_, src);                         \
                                                                        \
      if (!THCudaTensor_pointwiseApply2(state, self_, src, Tensor##NAME##Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING); \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(hipGetLastError());                                    \
  }

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log, log)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, log1p)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(exp, exp)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(cos, cos)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(acos, acos)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(cosh, cosh)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(sin, sin)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(asin, asin)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(sinh, sinh)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(tan, tan)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(atan, atan)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(tanh, tanh)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(sqrt, sqrt)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(ceil, ceil)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(floor, floor)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(abs, fabs)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(round, roundf)

#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC