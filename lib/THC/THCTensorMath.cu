#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCTensorRandom.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

struct TensorFillOp {
  TensorFillOp(float v) : val(v) {}
  __device__ __forceinline__ void operator()(float* v) { *v = val; }

  const float val;
};

void THCudaTensor_fill(THCState* state, THCudaTensor *self_, float value)
{
  if (!THCudaTensor_pointwiseApply1(state, self_, TensorFillOp(value))) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

void THCudaTensor_zero(THCState *state, THCudaTensor *self_)
{
  if (THCudaTensor_isContiguous(state, self_)) {
    THCudaCheck(hipMemsetAsync(THCudaTensor_data(state, self_),
                                0,
                                sizeof(float) * THCudaTensor_nElement(state, self_)));
  } else {
    if (!THCudaTensor_pointwiseApply1(state, self_, TensorFillOp(0))) {
      THArgCheck(false, 1, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

void THCudaTensor_zeros(THCState *state, THCudaTensor *r_, THLongStorage *size)
{
  THCudaTensor_resize(state, r_, size, NULL);
  THCudaTensor_zero(state, r_);
}

void THCudaTensor_ones(THCState *state, THCudaTensor *r_, THLongStorage *size)
{
  THCudaTensor_resize(state, r_, size, NULL);
  THCudaTensor_fill(state, r_, 1);
}

void THCudaTensor_reshape(THCState *state, THCudaTensor *r_, THCudaTensor *t, THLongStorage *size)
{
  THCudaTensor_resize(state, r_, size, NULL);
  THCudaTensor_copy(state, r_, t);
}

long THCudaTensor_numel(THCState *state, THCudaTensor *t)
{
  return THCudaTensor_nElement(state, t);
}

struct TensorAddOp {
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out += *in;
  }

  __device__ __forceinline__ void operator()(float* out, float* in1, float* in2) {
    *out = *in1 + *in2;
  }
};

struct TensorCAddOp {
  TensorCAddOp(float v) : val(v) {}

  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out += val * *in;
  }

  __device__ __forceinline__ void operator()(float* out, float* in1, float* in2) {
    *out = *in1 + val * *in2;
  }

  float val;
};

void THCudaTensor_cadd(THCState *state, THCudaTensor *self_, THCudaTensor* src1, float value, THCudaTensor *src2)
{
  THArgCheck(THCudaTensor_nElement(state, src1) ==
             THCudaTensor_nElement(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    if (value == 1.0f) {
      // self += src2
      if (!THCudaTensor_pointwiseApply2(state, self_, src2, TensorAddOp())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self += value * src2
      if (!THCudaTensor_pointwiseApply2(state, self_, src2, TensorCAddOp(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src1);

    if (value == 1.0f) {
      // self = src1 + src2
      if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorAddOp())) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    } else {
      // self = src1 + value * src2
      if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorCAddOp(value))) {
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);
      }
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorMulOp {
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out *= *in;
  }

  __device__ __forceinline__ void operator()(float* out, float* in1, float* in2) {
    *out = *in1 * *in2;
  }
};

void THCudaTensor_cmul(THCState *state, THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THArgCheck(THCudaTensor_nElement(state, src1) ==
             THCudaTensor_nElement(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self *= src2
    if (!THCudaTensor_pointwiseApply2(state, self_, src2, TensorMulOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src1);

    // self = src1 * src2
    if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorMulOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorCPowOp {
  __device__ __forceinline__ void operator()(float* out, float* in) {
    *out = powf(*out, *in);
  }

  __device__ __forceinline__ void operator()(float* out, float* in1, float* in2) {
    *out = powf(*in1, *in2);
  }
};

void THCudaTensor_cpow(THCState *state, THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THArgCheck(THCudaTensor_nElement(state, src1) ==
             THCudaTensor_nElement(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self = pow(self, src2)
    if (!THCudaTensor_pointwiseApply2(state, self_, src2, TensorCPowOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src1);

    // self = pow(src1, src2)
    if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorCPowOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorDivOp {
  __device__ __forceinline__ void
  operator()(float* out, float* in) {
    *out /= *in;
  }

  __device__ __forceinline__ void
  operator()(float* out, float* in1, float* in2) {
    *out = *in1 / *in2;
  }
};

void THCudaTensor_cdiv(THCState* state, THCudaTensor *self_, THCudaTensor *src1, THCudaTensor *src2)
{
  THArgCheck(THCudaTensor_nElement(state, src1) ==
             THCudaTensor_nElement(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self *= src2
    if (!THCudaTensor_pointwiseApply2(state, self_, src2, TensorDivOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src1);

    // self = src1 * src2
    if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorDivOp())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorAddCMulOp {
  TensorAddCMulOp(float v) : val(v) {}

  __device__ __forceinline__ void
  operator()(float* out, float* in1, float* in2) {
    *out += val * *in1 * *in2;
  }

  float val;
};

void THCudaTensor_addcmul(THCState *state, THCudaTensor *self_, THCudaTensor *t, float value, THCudaTensor *src1, THCudaTensor *src2)
{
  if(self_ != t)
  {
    THCudaTensor_resizeAs(state, self_, t);
    THCudaTensor_copy(state, self_, t);
  }
  THCudaTensor_resizeAs(state, self_, src1);

  THArgCheck(THCudaTensor_nElement(state, src1) ==
             THCudaTensor_nElement(state, src2), 3, "sizes do not match");

  if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorAddCMulOp(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

struct TensorAddCDivOp {
  TensorAddCDivOp(float v) : val(v) {}

  __device__ __forceinline__ void
  operator()(float* out, float* in1, float* in2) {
    *out += val * *in1 / *in2;
  }

  float val;
};

void THCudaTensor_addcdiv(THCState *state, THCudaTensor *self_, THCudaTensor *t, float value, THCudaTensor *src1, THCudaTensor *src2)
{
  if(self_ != t)
  {
    THCudaTensor_resizeAs(state, self_, t);
    THCudaTensor_copy(state, self_, t);
  }

  THCudaTensor_resizeAs(state, self_, src1);
  THArgCheck(THCudaTensor_nElement(state, src1) == THCudaTensor_nElement(state, src2), 3, "sizes do not match");

  if (!THCudaTensor_pointwiseApply3(state, self_, src1, src2, TensorAddCDivOp(value))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

float THCudaTensor_minall(THCState *state, THCudaTensor *self)
{
  self = THCudaTensor_newContiguous(state, self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(state, self));

  float result = thrust::reduce(self_data, self_data+THCudaTensor_nElement(state, self), (float)(THInf), thrust::minimum<float>());

  THCudaTensor_free(state, self);
  return result;
}

float THCudaTensor_maxall(THCState *state, THCudaTensor *self)
{
  self = THCudaTensor_newContiguous(state, self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(state, self));

  float result = thrust::reduce(self_data, self_data+THCudaTensor_nElement(state, self), (float)(-THInf), thrust::maximum<float>());

  THCudaTensor_free(state, self);
  return result;
}

float THCudaTensor_sumall(THCState *state, THCudaTensor *self)
{
  self = THCudaTensor_newContiguous(state, self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(state, self));

  float result = thrust::reduce(self_data, self_data+THCudaTensor_nElement(state, self), (float)(0), thrust::plus<float>());

  THCudaTensor_free(state, self);
  return result;
}

float THCudaTensor_prodall(THCState *state, THCudaTensor *self)
{
  self = THCudaTensor_newContiguous(state, self);
  thrust::device_ptr<float> self_data(THCudaTensor_data(state, self));

  float result = thrust::reduce(self_data, self_data+THCudaTensor_nElement(state, self), (float)(1), thrust::multiplies<float>());

  THCudaTensor_free(state, self);
  return result;
}

struct dim4 {
    unsigned arr[4];

    __host__ dim4(unsigned init=0) {
        for(unsigned i=0; i<4; i++) { arr[i] = init; }
    }

    __host__ __device__ unsigned& operator[](const unsigned& idx) { return arr[idx]; }
};

void THCudaTensor_sum(THCState* state, THCudaTensor *self, THCudaTensor *src, long dimension)
{
  THCudaTensor_reduceDim(
    state, self, src,
    thrust::identity<float>(), thrust::plus<float>(), 0.0f, dimension);

  THCudaCheck(hipGetLastError());
}

void THCudaTensor_prod(THCState* state, THCudaTensor *self, THCudaTensor *src, long dimension)
{
  THCudaTensor_reduceDim(
    state, self, src,
    thrust::identity<float>(), thrust::multiplies<float>(), 1.0f, dimension);

  THCudaCheck(hipGetLastError());
}
